#include "hip/hip_runtime.h"
#include "2DKerns.cuh"
//#include <math>
#include "hip/hip_math_constants.h"
#include "windows.h"

#define KERNEL_RADIUS 4
#define TILE_W 16

__global__ void kernel(unsigned short* input_image, unsigned short* output_image, int width, int height, float* d_Kernel, int kernSize)
{
		int index_x = blockIdx.x*blockDim.x + threadIdx.x;
		int index_y = blockIdx.y*blockDim.y + threadIdx.y;

		//map the two 2D indices to a single linear 1D index
		int grid_width = gridDim.x*blockDim.x;
		int index = index_y*grid_width + index_x;

		unsigned short value;
		long sum = 0;
		
		int kernRad = kernSize/2;
		for (int i = -kernRad; i <= kernRad; i++)
		{
			for (int j = -kernRad; j <= kernRad; j++)	// col wise
			{
				// check row first
				if (blockIdx.x == 0 && (threadIdx.x + i) < 0)	// left apron
					value = 0;
				else if ( blockIdx.x == (gridDim.x - 1) && (threadIdx.x + i) > blockDim.x-1 )	// right apron
					value = 0;
				else 
				{ 
					// check col next
					if (blockIdx.y == 0 && (threadIdx.y + j) < 0)	// top apron
						value = 0;
					else if ( blockIdx.y == (gridDim.y - 1) &&
						(threadIdx.y + j) > blockDim.y-1 )	// bottom apron
						value = 0;
					else	// safe case
						value = input_image[index + i + j * width];
				} 
				sum += value * d_Kernel[kernRad + i] * d_Kernel[kernRad + j];
			}
		}
		output_image[index] = sum;
	
}

__global__ void kernelShared(unsigned short* input_image, unsigned short* output_image, int width, int height, float* d_Kernel)
{
	int index_x = blockIdx.x*blockDim.x + threadIdx.x;
	int index_y = blockIdx.y*blockDim.y + threadIdx.y;
	//map the two 2D indices to a single linear 1D index
	int grid_width = gridDim.x*blockDim.x;
	int index = index_y*grid_width + index_x;

		
	__shared__ float cache[TILE_W+(2*KERNEL_RADIUS)][TILE_W+(2*KERNEL_RADIUS)];

	int x = index_x-KERNEL_RADIUS;
	int y = index_y-KERNEL_RADIUS;
	if( x < 0 || y < 0)
		cache[threadIdx.x][threadIdx.y] = 0;
	else
		cache[threadIdx.x][threadIdx.y] = input_image[index-KERNEL_RADIUS-width*KERNEL_RADIUS];

	x = index_x+KERNEL_RADIUS;
	y = index_y-KERNEL_RADIUS;
	if( x >= width-1 || y < 0)
		cache[threadIdx.x + blockDim.x][threadIdx.y] = 0;
	else
		cache[threadIdx.x + blockDim.x][threadIdx.y] = input_image[index+KERNEL_RADIUS-width*KERNEL_RADIUS];
		


	x = index_x-KERNEL_RADIUS;
	y = index_y+KERNEL_RADIUS;

	if( x < 0 || y >= height)
		cache[threadIdx.x][threadIdx.y + blockDim.y] = 0;
	else
		cache[threadIdx.x][threadIdx.y + blockDim.y] = input_image[index-KERNEL_RADIUS+width*KERNEL_RADIUS];
		
	x = index_x+KERNEL_RADIUS;

	y = index_y+KERNEL_RADIUS;
	if( x >= width || y >= height)
		cache[threadIdx.x + blockDim.x][threadIdx.y + blockDim.y] = 0;
	else
		cache[threadIdx.x + blockDim.x][threadIdx.y + blockDim.y] = input_image[index+KERNEL_RADIUS+width*KERNEL_RADIUS];
		
	__syncthreads();
	//output_image[index] = input_image[index];

	float sum = 0.0;

	x = KERNEL_RADIUS + threadIdx.x;
	y = KERNEL_RADIUS + threadIdx.y;
	for(int i = -KERNEL_RADIUS;i<=KERNEL_RADIUS;++i)
	{
		sum += cache[x+i][y]*d_Kernel[KERNEL_RADIUS+i];
	}
	for(int j = -KERNEL_RADIUS;j<=KERNEL_RADIUS;++j)
	{
		sum += cache[x][y+j]*d_Kernel[KERNEL_RADIUS+j];
	}
	/*for(int i = -KERNEL_RADIUS; i<=KERNEL_RADIUS; ++i)
	{
		for(int j = -KERNEL_RADIUS; i<=KERNEL_RADIUS; ++j)
		{
			//sum += cache[x+i][y+j]*d_Kernel[KERNEL_RADIUS+i]*d_Kernel[KERNEL_RADIUS+j];
			sum += 4500*d_Kernel[KERNEL_RADIUS+j];
		}
	}*/
	sum /=2;
	output_image[index] = unsigned short(sum);	
}

__global__ void kernelSharedCustom(unsigned short* input_image, unsigned short* output_image, int width, int height, float* d_Kernel)
{
	int x;
	int y;
	int index_x = blockIdx.x*blockDim.x + threadIdx.x;
	int index_y = blockIdx.y*blockDim.y + threadIdx.y;
	//map the two 2D indices to a single linear 1D index
	int grid_width = gridDim.x*blockDim.x;
	int index = index_y*grid_width + index_x;

	__shared__ float cache[TILE_W+(2*KERNEL_RADIUS)][TILE_W];
	__shared__ float cache2[TILE_W][TILE_W+(2*KERNEL_RADIUS)];

	
	cache[threadIdx.x+KERNEL_RADIUS][threadIdx.y] = input_image[index];
	cache2[threadIdx.x][threadIdx.y+KERNEL_RADIUS] = input_image[index];

	
	x = index_x-KERNEL_RADIUS;		
	if(threadIdx.x<KERNEL_RADIUS)
	{
		if(x<0)
			cache[threadIdx.x][threadIdx.y]=0;
		else
			cache[threadIdx.x][threadIdx.y] = input_image[index-KERNEL_RADIUS];
	}		
	x = index_x+KERNEL_RADIUS;
	if(threadIdx.x>=(blockDim.x-KERNEL_RADIUS))
	{
		if(x>=width)
			cache[threadIdx.x+2*KERNEL_RADIUS][threadIdx.y]=0;
		else
			cache[threadIdx.x+2*KERNEL_RADIUS][threadIdx.y]= input_image[index+KERNEL_RADIUS];
	}
	
	y = index_y-KERNEL_RADIUS;
	if(threadIdx.y<KERNEL_RADIUS)
	{
		if(y<0)
			cache2[threadIdx.x][threadIdx.y]=0;
		else
			cache2[threadIdx.x][threadIdx.y] = input_image[index-width*KERNEL_RADIUS];
	}
	y=index_y+KERNEL_RADIUS;
	if(threadIdx.y>=(blockDim.y-KERNEL_RADIUS))
	{
		if(y>=height)
			cache2[threadIdx.x][threadIdx.y+2*KERNEL_RADIUS] = 0;
		else
			cache2[threadIdx.x][threadIdx.y+2*KERNEL_RADIUS] = input_image[index+width*KERNEL_RADIUS];
	}

	__syncthreads();
	
	//output_image[index] = input_image[index];
	
	float sum = 0.0;

	x = KERNEL_RADIUS + threadIdx.x;
	y = threadIdx.y;//KERNEL_RADIUS + threadIdx.y;
	for(int i = -KERNEL_RADIUS;i<=KERNEL_RADIUS;++i)
	{
		sum += cache[x+i][y]*d_Kernel[KERNEL_RADIUS+i];
	}		
	x = threadIdx.x;
	y = threadIdx.y+KERNEL_RADIUS;
	for(int j = -KERNEL_RADIUS;j<=KERNEL_RADIUS;++j)
	{
		sum += cache2[x][y+j]*d_Kernel[KERNEL_RADIUS+j];
	}
	sum /=2;
	output_image[index] = unsigned short(sum);
}

__global__ void kernelSharedCustom2(unsigned short* input_image, unsigned short* output_image, int width, int height, float* d_Kernel)
{
	int x;
	int y;
	int index_x = blockIdx.x*blockDim.x + threadIdx.x;
	int index_y = blockIdx.y*blockDim.y + threadIdx.y;
	//map the two 2D indices to a single linear 1D index
	int grid_width = gridDim.x*blockDim.x;
	int index = index_y*grid_width + index_x;

	__shared__ float cache[TILE_W+(2*KERNEL_RADIUS)][TILE_W+(2*KERNEL_RADIUS)];
	
	cache[threadIdx.x+KERNEL_RADIUS][threadIdx.y+KERNEL_RADIUS] = input_image[index];
	
	if(threadIdx.x<KERNEL_RADIUS)
	{
		x = index_x-KERNEL_RADIUS;
		if (threadIdx.y<KERNEL_RADIUS)
		{
			y = index_y-KERNEL_RADIUS;
			if(x<0 || y<0)
				cache[threadIdx.x][threadIdx.y]=0;
			else
				cache[threadIdx.x][threadIdx.y] = input_image[index-KERNEL_RADIUS - width*KERNEL_RADIUS];
		}
		if (threadIdx.y>=(blockDim.y-KERNEL_RADIUS))
		{
			y = index_y+KERNEL_RADIUS;
			if(x<0 || y>=height)
				cache[threadIdx.x][threadIdx.y+2*KERNEL_RADIUS]=0;
			else
				cache[threadIdx.x][threadIdx.y+2*KERNEL_RADIUS] = input_image[index-KERNEL_RADIUS + width*KERNEL_RADIUS];
		}
		if(x<0)
			cache[threadIdx.x][threadIdx.y+KERNEL_RADIUS]=0;
		else
			cache[threadIdx.x][threadIdx.y+KERNEL_RADIUS] = input_image[index-KERNEL_RADIUS];
	}		
	if(threadIdx.x>=(blockDim.x-KERNEL_RADIUS))
	{
		x = index_x+KERNEL_RADIUS;

		if(threadIdx.y<KERNEL_RADIUS)
		{
			y = index_y-KERNEL_RADIUS;
			if(x>=width || y<0)
				cache[threadIdx.x+2*KERNEL_RADIUS][threadIdx.y]=0;
			else
				cache[threadIdx.x+2*KERNEL_RADIUS][threadIdx.y]=input_image[index+KERNEL_RADIUS-width*KERNEL_RADIUS];
		}
		if (threadIdx.y>=height)
		{
			y = index_y+KERNEL_RADIUS;
			if(x>=width || y>=(blockDim.x-KERNEL_RADIUS))
				cache[threadIdx.x+2*KERNEL_RADIUS][threadIdx.y+2*KERNEL_RADIUS]=0;
			else
				cache[threadIdx.x+2*KERNEL_RADIUS][threadIdx.y+2*KERNEL_RADIUS] = input_image[index+KERNEL_RADIUS + width*KERNEL_RADIUS];
		}		
		
		if(x>=width)
			cache[threadIdx.x+2*KERNEL_RADIUS][threadIdx.y+KERNEL_RADIUS]=0;
		else
			cache[threadIdx.x+2*KERNEL_RADIUS][threadIdx.y+KERNEL_RADIUS]= input_image[index+KERNEL_RADIUS];
		
	}
	if(threadIdx.y<KERNEL_RADIUS)
	{
		y = index_y-KERNEL_RADIUS;
		if(y<0)
			cache[threadIdx.x+KERNEL_RADIUS][threadIdx.y]=0;
		else
			cache[threadIdx.x+KERNEL_RADIUS][threadIdx.y] = input_image[index-width*KERNEL_RADIUS];
	}	
	if(threadIdx.y>=(blockDim.y-KERNEL_RADIUS))
	{
		y=index_y+KERNEL_RADIUS;
		if(y>=height)
			cache[threadIdx.x+KERNEL_RADIUS][threadIdx.y+2*KERNEL_RADIUS] = 0;
		else
			cache[threadIdx.x+KERNEL_RADIUS][threadIdx.y+2*KERNEL_RADIUS] = input_image[index+width*KERNEL_RADIUS];
	}

	__syncthreads();
	
	//output_image[index] = input_image[index];
	
	float sum = 0.0;

	x = KERNEL_RADIUS + threadIdx.x;
	y = KERNEL_RADIUS + threadIdx.y;
	for(int i = -KERNEL_RADIUS;i<=KERNEL_RADIUS;++i)
	{
		sum += cache[x+i][y]*d_Kernel[KERNEL_RADIUS+i];
	}		
	for(int j = -KERNEL_RADIUS;j<=KERNEL_RADIUS;++j)
	{
		sum += cache[x][y+j]*d_Kernel[KERNEL_RADIUS+j];
	}
	sum /=2;
	output_image[index] = unsigned short(sum);
}

__global__ void kernelSharedCustomDynamic(unsigned short* input_image, unsigned short* output_image, int width, int height, float* d_Kernel, int kernRadius)
{
	int x;
	int y;
	int index_x = blockIdx.x*blockDim.x + threadIdx.x;
	int index_y = blockIdx.y*blockDim.y + threadIdx.y;
	//map the two 2D indices to a single linear 1D index
	int grid_width = gridDim.x*blockDim.x;
	int index = index_y*grid_width + index_x;
	int cacheWidth = 2*kernRadius+blockDim.x;

	extern __shared__ float cache[];
	cache[threadIdx.x+KERNEL_RADIUS + cacheWidth*(threadIdx.y+KERNEL_RADIUS)] = input_image[index];
	
	
	if(threadIdx.x<KERNEL_RADIUS)
	{
		x = index_x-KERNEL_RADIUS;
		if (threadIdx.y<KERNEL_RADIUS)
		{
			y = index_y-KERNEL_RADIUS;
			if(x<0 || y<0)
				cache[threadIdx.x+cacheWidth*threadIdx.y]=0;
			else
				cache[threadIdx.x+cacheWidth*threadIdx.y] = input_image[index-KERNEL_RADIUS - width*KERNEL_RADIUS];
		}
		if (threadIdx.y>=(blockDim.y-KERNEL_RADIUS))
		{
			y = index_y+KERNEL_RADIUS;
			if(x<0 || y>=height)
				cache[threadIdx.x+cacheWidth*(threadIdx.y+2*KERNEL_RADIUS)]=0;
			else
				cache[threadIdx.x+cacheWidth*(threadIdx.y+2*KERNEL_RADIUS)] = input_image[index-KERNEL_RADIUS + width*KERNEL_RADIUS];
		}
		if(x<0)
			cache[threadIdx.x+cacheWidth*(threadIdx.y+KERNEL_RADIUS)]=0;
		else
			cache[threadIdx.x+cacheWidth*(threadIdx.y+KERNEL_RADIUS)] = input_image[index-KERNEL_RADIUS];
	}		
	if(threadIdx.x>=(blockDim.x-KERNEL_RADIUS))
	{
		x = index_x+KERNEL_RADIUS;

		if(threadIdx.y<KERNEL_RADIUS)
		{
			y = index_y-KERNEL_RADIUS;
			if(x>=width || y<0)
				cache[threadIdx.x+2*KERNEL_RADIUS+cacheWidth*threadIdx.y]=0;
			else
				cache[threadIdx.x+2*KERNEL_RADIUS+cacheWidth*threadIdx.y]=input_image[index+KERNEL_RADIUS-width*KERNEL_RADIUS];
		}
		if (threadIdx.y>=(blockDim.y-KERNEL_RADIUS))
		{
			y = index_y+KERNEL_RADIUS;
			if(x>=width || y>=height)
				cache[threadIdx.x+2*KERNEL_RADIUS+cacheWidth*(threadIdx.y+2*KERNEL_RADIUS)]=0;
			else
				cache[threadIdx.x+2*KERNEL_RADIUS+cacheWidth*(threadIdx.y+2*KERNEL_RADIUS)] = input_image[index+KERNEL_RADIUS + width*KERNEL_RADIUS];
		}		
		
		if(x>=width)
			cache[threadIdx.x+2*KERNEL_RADIUS+cacheWidth*(threadIdx.y+KERNEL_RADIUS)]=0;
		else
			cache[threadIdx.x+2*KERNEL_RADIUS+cacheWidth*(threadIdx.y+KERNEL_RADIUS)]= input_image[index+KERNEL_RADIUS];
		
	}
	if(threadIdx.y<KERNEL_RADIUS)
	{
		y = index_y-KERNEL_RADIUS;
		if(y<0)
			cache[threadIdx.x+KERNEL_RADIUS+cacheWidth*threadIdx.y]=0;
		else
			cache[threadIdx.x+KERNEL_RADIUS+cacheWidth*threadIdx.y] = input_image[index-width*KERNEL_RADIUS];
	}	
	if(threadIdx.y>=(blockDim.y-KERNEL_RADIUS))
	{
		y=index_y+KERNEL_RADIUS;
		if(y>=height)
			cache[threadIdx.x+KERNEL_RADIUS+cacheWidth*(threadIdx.y+2*KERNEL_RADIUS)] = 0;
		else
			cache[threadIdx.x+KERNEL_RADIUS+cacheWidth*(threadIdx.y+2*KERNEL_RADIUS)] = input_image[index+width*KERNEL_RADIUS];
	}

	__syncthreads();
	
	//output_image[index] = input_image[index];
	
	float sum = 0.0;

	x = KERNEL_RADIUS + threadIdx.x;
	y = KERNEL_RADIUS + threadIdx.y;
	for(int i = -KERNEL_RADIUS;i<=KERNEL_RADIUS;++i)
	{
		sum += cache[x+i+cacheWidth*y]*d_Kernel[KERNEL_RADIUS+i];
	}		
	for(int j = -KERNEL_RADIUS;j<=KERNEL_RADIUS;++j)
	{
		sum += cache[x+cacheWidth*(y+j)]*d_Kernel[KERNEL_RADIUS+j];
	}
	sum /=2;
	output_image[index] = unsigned short(sum);
}

int main(void)
{
	LARGE_INTEGER frequency;
	LARGE_INTEGER t1, t2, t3, t4;
	double elapsedTime;

	unsigned short num_elements_x = 1536;
	
	unsigned short num_elements_y = 1536;
	int kernSize = 2*KERNEL_RADIUS+1;
	float sigma = 10.0;


	int num_bytes = num_elements_x*num_elements_y*sizeof(unsigned short);
	int kern_bytes = kernSize*sizeof(float);

	FILE *fin = fopen("input.raw","r");
	if(fin==NULL)
	{
		printf("Could Not Find Dark Image!\n");
		return -1;
	}

	unsigned short* host_array = (unsigned short*)malloc(num_bytes);
	unsigned short* host_array2 = (unsigned short*)malloc(num_bytes);
	unsigned short* host_array3 = (unsigned short*)malloc(num_bytes);
	float* host_kern = (float*)malloc(kern_bytes);
	
	float sum = 0.0;
	for(int i = 0; i<kernSize; i++)
	{
		int x = i-kernSize/2;
		float temp = 1/(sqrt(2*HIP_PI_F *sigma))*exp((-1*x*x)/(2*sigma*sigma));
		sum += temp;
		host_kern[i] = temp;
		//printf("kernel at %d (x = %d): %f\n", i, x, temp);
	}

	for(int i = 0; i<kernSize; i++)
	{
		host_kern[i] /= sum;
	}

	
	size_t read =fread(host_array,num_bytes,1,fin);
	fclose(fin);
	printf("Image Loaded...\n");

	unsigned short* device_array_in = 0;
	unsigned short* device_array_out = 0;
	float* device_kern = 0;

	//allocate memory in either space
	hipMalloc((void**)&device_array_in,num_bytes);
	hipMalloc((void**)&device_array_out,num_bytes);
	hipMalloc((void**)&device_kern,kern_bytes);

	unsigned short* device_array_in2 = 0;
	unsigned short* device_array_out2 = 0;
	float* device_kern2 = 0;
	hipMalloc((void**)&device_array_in2,num_bytes);
	hipMalloc((void**)&device_array_out2,num_bytes);
	hipMalloc((void**)&device_kern2,kern_bytes);

	hipMemcpy(device_array_in,host_array,num_bytes,hipMemcpyHostToDevice);
	hipMemcpy(device_kern,host_kern,kern_bytes, hipMemcpyHostToDevice);
	hipMemcpy(device_array_in2,host_array,num_bytes,hipMemcpyHostToDevice);
	hipMemcpy(device_kern2,host_kern,kern_bytes, hipMemcpyHostToDevice);
	
	
	//create two dimensional 4x4 thread blocks
	dim3 block_size;
	block_size.x = TILE_W;
	block_size.y = TILE_W;

	//configure a two dimensional grid as well
	dim3 grid_size;
	grid_size.x = num_elements_x/block_size.x;
	grid_size.y = num_elements_y/block_size.y;

	//grid_size & block_size are passed as arguments to the triple chevrons
	/*int sharedMemory = (block_size.x+2*KERNEL_RADIUS)*(block_size.y+2*KERNEL_RADIUS)*sizeof(float);
	kernelSharedCustomDynamic<<<grid_size,block_size,sharedMemory>>>(device_array_in,device_array_out,num_elements_x,num_elements_y,device_kern,KERNEL_RADIUS);
	printf("Cuda error: %s\n", hipGetErrorString(hipGetLastError()));
	hipMemcpy(host_array2,device_array_out,num_bytes,hipMemcpyDeviceToHost);
	FILE* fout = fopen("Dynamic-output.raw","wb");
	int written = fwrite(host_array2,sizeof(unsigned short),num_bytes/2,fout);
	fclose(fout);*/

	/*
	kernelSharedCustom2<<<grid_size,block_size>>>(device_array_in,device_array_out,num_elements_x,num_elements_y,device_kern);
	printf("Cuda error: %s\n", hipGetErrorString(hipGetLastError()));
	hipMemcpy(host_array2,device_array_out,num_bytes,hipMemcpyDeviceToHost);
	FILE* fout = fopen("Custom-SingleCache-output.raw","wb");
	int written = fwrite(host_array2,sizeof(unsigned short),num_bytes/2,fout);
	fclose(fout);*/

	/*
	kernelSharedCustom<<<grid_size,block_size>>>(device_array_in,device_array_out,num_elements_x,num_elements_y,device_kern);
	printf("Cuda error: %s\n", hipGetErrorString(hipGetLastError()));
	hipMemcpy(host_array2,device_array_out,num_bytes,hipMemcpyDeviceToHost);
	FILE* fout = fopen("Custom-output.raw","wb");
	int written = fwrite(host_array2,sizeof(unsigned short),num_bytes/2,fout);
	fclose(fout);*/

	/*
	kernelShared<<<grid_size,block_size>>>(device_array_in,device_array_out,num_elements_x,num_elements_y,device_kern);
	printf("Cuda error: %s\n", hipGetErrorString(hipGetLastError()));
	hipMemcpy(host_array2,device_array_out,num_bytes,hipMemcpyDeviceToHost);
	FILE* fout = fopen("Standard-output.raw","wb");
	int written = fwrite(host_array2,sizeof(unsigned short),num_bytes/2,fout);
	fclose(fout);*/

	
	//grid_size & block_size are passed as arguments to the triple chevrons
	int numIts = 50;
	double sum3 = 0.0;
	double sum2 = 0.0;
	int sharedMemory = (block_size.x+2*KERNEL_RADIUS)*(block_size.y+2*KERNEL_RADIUS)*sizeof(float);
	QueryPerformanceFrequency(&frequency);
	for(int i = 0; i< numIts;i++)
	{
		printf("Dynamic %d\n",i);
		QueryPerformanceCounter(&t1);
		//kernelSharedCustom<<<grid_size,block_size>>>(device_array_in,device_array_out,num_elements_x,num_elements_y,device_kern);
		kernelSharedCustomDynamic<<<grid_size,block_size,sharedMemory>>>(device_array_in,device_array_out,num_elements_x,num_elements_y,device_kern,KERNEL_RADIUS);
		QueryPerformanceCounter(&t2);
		sum3 += (t2.QuadPart - t1.QuadPart) * 1000.0/ frequency.QuadPart;
	}
	hipMemcpy(host_array2,device_array_out,num_bytes,hipMemcpyDeviceToHost);

	for(int i = 0; i< numIts; i++)
	{
		printf("Single-Cache %d\n",i);
		QueryPerformanceCounter(&t1);
		kernelSharedCustom2<<<grid_size,block_size>>>(device_array_in2,device_array_out2,num_elements_x,num_elements_y,device_kern2);
		QueryPerformanceCounter(&t2);
		sum2 += (t2.QuadPart - t1.QuadPart) * 1000.0/ frequency.QuadPart;
	}
	
	hipMemcpy(host_array3,device_array_out2,num_bytes,hipMemcpyDeviceToHost);

	sum2 /= numIts;
	sum3 /= numIts;

	printf("Single-Cache processing average time: %f ms\n",sum2);
	printf("Dynamic processing average time: %f ms\n",sum3);
	
	printf("Cuda error: %s\n", hipGetErrorString(hipGetLastError()));

	printf("Image downloaded from device!\n");

	FILE* fout = fopen("Custom-Dynamic-output.raw","wb");
	int written = fwrite(host_array2,sizeof(unsigned short),num_bytes/2,fout);
	fclose(fout);

	FILE* fout2 = fopen("Custom-Single-output.raw","wb");
	written = fwrite(host_array3,sizeof(unsigned short),num_bytes/2,fout2);
	fclose(fout2);
	

	printf("\n");

	//deallocate memory
	free(host_array);
	hipFree(device_array_in);
	hipFree(device_array_out);
	hipFree(device_array_in2);
	hipFree(device_array_out2);
	hipFree(device_kern);
	hipFree(device_kern2);
	return 0;
}